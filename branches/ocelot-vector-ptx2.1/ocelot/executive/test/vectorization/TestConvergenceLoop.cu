/*!
	\file TestConvergenceLoop.cu
	\date 17 February 2011
	\author Andrew Kerr <arkerr@gatech.edu>
	\brief Demonstrates convergent execution
*/

// Standard Library Includes

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

/////////////////////////////////////////////////////////////////////////////////////////////////

/*!

*/
extern "C" __global__ void convergenceWithLoop(float *A, int P) {
	int i = threadIdx.x;
	float f = i * 2.0f + 1.0f;;
	for (int p = 0; p < P; p++) {
		f *= 1.25f - 2.0f;
	}
	A[i] = f;
}

/*!
	\brief loop with divergent control flow
*/
extern "C" __global__ void loopEarlyExit(float *A, float dt) {
	int i = threadIdx.x;
	float f = A[i];
	for (int p = 0; p < i + 1; p++) {
		f += dt;
	}
	A[i] = f;
}

static float expectedValue(int n,  int P) {
	int i = n;
	float f = i * 2.0f + 1.0f;;
	for (int p = 0; p < P; p++) {
		f *= 1.25f - 2.0f;
	}
	return f;
}

static int testConvergenceWithLoop() {
	const int N = 4;
	const int P = 3;
	float *A_host, *A_device;
	size_t bytes = N * sizeof(float);
	
	A_host = (float *)malloc(bytes);
	hipMalloc((void **)&A_device, bytes);
	
	for (int i = 0; i < N; i++) {
		A_host[i] = (float)(i+1);
	}
	hipMemcpy(A_device, A_host, bytes, hipMemcpyHostToDevice);
	
	convergenceWithLoop<<< dim3(1,1), dim3(N, 1, 1) >>>(A_device, P);
	
	hipMemcpy(A_host, A_device, bytes, hipMemcpyDeviceToHost);
	
	int errors = 0;
	for (int i = 0; (errors < 5) && i < N; i++) {
		float expected = expectedValue(i, P);
		float got = A_host[i];
		if (fabs(expected - got) > 0.001f) {
			++errors;
			printf("error 1 [%d] - expected: %f, got %f\n", i, expected, got);
		}
	}
	
	hipFree(A_device);
	free(A_host);
	return errors;
}

static int testLoopEarlyExit() {
	const int N = 8;
	float *A_host, *A_gpu;
	
	size_t bytes = sizeof(float)*N;
	hipMalloc((void **)&A_gpu, bytes);
	A_host = (float *)malloc(bytes);
	
	for (int i = 0; i < N; i++) {
		A_host[i] = 2.0f * (float)i / (float)(N-1);
	}
	
	hipMemcpy(A_gpu, A_host, bytes, hipMemcpyHostToDevice);
	
	float dt = 0.25f;
	dim3 grid(1,1);
	dim3 block(N,1,1);
	
	loopEarlyExit<<< grid, block >>>(A_gpu, dt);

	hipMemcpy(A_host, A_gpu, bytes, hipMemcpyDeviceToHost);
	int errors = 0;
	for (int i = 0; errors < 5 && i < N; i++) {
		float got = A_host[i];
		float expected = 2.0f * (float)i / (float)(N-1) + i * dt;
		printf("ERROR 2 [%d] - expected: %f, got: %f\n", i, expected, got);
		++errors;
	}
	
	free(A_host);
	hipFree(A_gpu);
	
	return errors;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main( int argc, char** argv )
{
	int errors = 0;
	if (!errors) {
		errors += testConvergenceWithLoop();
	}
	if (!errors) {
		errors += testLoopEarlyExit();
	}

	printf("Pass/Fail : %s\n", (errors ? "Fail":"Pass"));

	return 0;
}